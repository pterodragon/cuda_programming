
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef NDEBUG
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
#else
#define gpuErrchk(ans)                                                         \
  {}
#endif

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void matrix_mult_kernel(int *a, int *b, int *c, int m, int n,
                                   int k) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum = 0;
  if (col < n && row < m) {
    for (int i = 0; i < k; i++) {
      sum += a[row * k + i] * b[i * n + col];
    }
    c[row * n + col] = sum;
  }
}

void matrix_mult(int *a, int *b, int *c, int m, int n, int k) {
  constexpr int BLOCK_WIDTH = 16;
  int n_blocks = ceil(m / (float)BLOCK_WIDTH);
  dim3 dimGrid(n_blocks, n_blocks);
  dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

  hipError_t err = hipSuccess;
  printf("[matrix_mult] (%d, %d) * (%d, %d)\n", m, k, k, n);
  int size_A = m * k * sizeof(int);
  int size_B = k * n * sizeof(int);
  int size_C = m * n * sizeof(int);

  int *d_A, *d_B, *d_C;
  err = hipMalloc((void **)&d_A, size_A);
  gpuErrchk(err);
  err = hipMalloc((void **)&d_B, size_B);
  gpuErrchk(err);
  err = hipMalloc((void **)&d_C, size_C);
  gpuErrchk(err);
  err = hipMemcpy(d_A, a, size_A, hipMemcpyHostToDevice);
  gpuErrchk(err);
  err = hipMemcpy(d_B, b, size_B, hipMemcpyHostToDevice);
  gpuErrchk(err);

  matrix_mult_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, m, n, k);
  err = hipMemcpy(c, d_C, size_C, hipMemcpyDeviceToHost);
  gpuErrchk(err);
  err = hipFree(d_A);
  gpuErrchk(err);
  err = hipFree(d_B);
  gpuErrchk(err);
  err = hipFree(d_C);
  gpuErrchk(err);
  err = hipDeviceReset();
  gpuErrchk(err);
}
